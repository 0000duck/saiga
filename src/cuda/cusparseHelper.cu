#include "saiga/cuda/cusparseHelper.h"

#include "saiga/util/assert.h"

using std::cout;
using std::endl;

namespace CUDA {

hipsparseHandle_t cusparseHandle = 0;
hipblasHandle_t cublashandle = 0;

void initBLASSPARSE(){
    if(!isBLASSPARSEInitialized()){
        hipblasCreate(&cublashandle);
        hipsparseCreate(&cusparseHandle);
    }
}

void destroyBLASSPARSE(){
    if(isBLASSPARSEInitialized()){
        hipsparseDestroy(cusparseHandle);
        hipblasDestroy(cublashandle);
        cusparseHandle = 0;
        cublashandle = 0;
    }
}

bool isBLASSPARSEInitialized(){
    return cusparseHandle != 0;
}

extern void testCuBLAS();
extern void testCuSparse();
void runBLASSPARSETests(){
    testCuBLAS();
    testCuSparse();
}

}
