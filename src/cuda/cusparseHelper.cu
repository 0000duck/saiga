#include "saiga/cuda/cusparseHelper.h"

#include "saiga/util/assert.h"

using std::cout;
using std::endl;

namespace CUDA {

hipsparseHandle_t cusparseHandle = 0;
hipblasHandle_t cublashandle = 0;

void initBLASSPARSE(){
    hipblasCreate(&cublashandle);
    hipsparseCreate(&cusparseHandle);
}

void destroyBLASSPARSE(){
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublashandle);
}

extern void testCuBLAS();
extern void testCuSparse();
void runBLASSPARSETests(){
    testCuBLAS();
    testCuSparse();
}

}
