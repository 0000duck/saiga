#include "saiga/cuda/cudaTimer.h"
#include "saiga/util/assert.h"

using std::cout;
using std::endl;

namespace CUDA {

CudaScopedTimerPrint::CudaScopedTimerPrint(const std::string &name) : name(name){

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
}

CudaScopedTimerPrint::~CudaScopedTimerPrint(){
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time;
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    std::cout << name << " : " << time << "ms." << std::endl;
}


}
