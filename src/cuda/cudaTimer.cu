#include "saiga/cuda/cudaTimer.h"
#include "saiga/util/assert.h"

using std::cout;
using std::endl;

namespace CUDA {

CudaScopedTimer::CudaScopedTimer(float& time) : time(time){

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
}

CudaScopedTimer::~CudaScopedTimer(){
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}





CudaScopedTimerPrint::CudaScopedTimerPrint(const std::string &name) : name(name){

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
}

CudaScopedTimerPrint::~CudaScopedTimerPrint(){
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time;
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    std::cout << name << " : " << time << "ms." << std::endl;
}


}
