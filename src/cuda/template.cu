////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <iostream>

#include <hip/hip_runtime.h>

namespace CUDA{

__global__ void
testKernel(float *g_idata, float *g_odata)
{
    int tid = threadIdx.x;

    // write data to global memory
    g_odata[tid] = g_idata[tid]+5;
}


int test()
{

    unsigned int num_threads = 32;
    unsigned int mem_size = sizeof(float) * num_threads;

    // allocate host memory
    float *h_idata = (float *) malloc(mem_size);

    // initalize the memory
    for (unsigned int i = 0; i < num_threads; ++i)
    {
        h_idata[i] = (float) i;
    }

    // allocate device memory
    float *d_idata;
    hipMalloc((void **) &d_idata, mem_size);
    // copy host memory to device
    hipMemcpy(d_idata, h_idata, mem_size,hipMemcpyHostToDevice);

    // allocate device memory for result
    float *d_odata;
    hipMalloc((void **) &d_odata, mem_size);

    // setup execution parameters
    dim3  grid(1, 1, 1);
    dim3  threads(num_threads, 1, 1);

    // execute the kernel
    testKernel<<< grid, threads, mem_size >>>(d_idata, d_odata);


    // allocate mem for the result on host side
    float *h_odata = (float *) malloc(mem_size);
    // copy result from device to host
    hipMemcpy(h_odata, d_odata, sizeof(float) * num_threads,hipMemcpyDeviceToHost);

    for (unsigned int i = 0; i < num_threads; ++i)
    {
        std::cout<<h_odata[i]<<std::endl;
    }


    // cleanup memory
    free(h_idata);
    free(h_odata);
    hipFree(d_idata);
    hipFree(d_odata);

}

}
