#include "hip/hip_runtime.h"
﻿/**
 * Copyright (c) 2017 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "saiga/cuda/imageProcessing/imageProcessing.h"
#include "saiga/cuda/device_helper.h"
#include "saiga/cuda/shfl_helper.h"

using std::cout;
using std::endl;

namespace Saiga {
namespace CUDA {


__constant__ float d_Kernel[SAIGA_MAX_KERNEL_SIZE];


template<typename T, int RADIUS, unsigned int BLOCK_W, unsigned int BLOCK_H, unsigned int Y_ELEMENTS>
__global__ static
void d_convolveOuterLinear(ImageView<T> src, ImageView<T> dst)
{
    const unsigned BLOCK_H2 = BLOCK_H * Y_ELEMENTS;

    //for radius = 4: elements = (32+8) * (16+8) = 960 = 3840
    __shared__ T buffer[BLOCK_H2 + 2*RADIUS][BLOCK_W + 2*RADIUS];
    //for radius = 4: elements = (32+8) * (16) = 640 = 2560
    __shared__ T buffer2[BLOCK_H2][BLOCK_W + 2*RADIUS];
    //total s mem per block = 6400
    //with 512 threads per block smem per sm: 25600 -> 100% occ


    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int t = tx + ty * BLOCK_W;
    int xp = blockIdx.x*BLOCK_W + tx;
    int yp = blockIdx.y*BLOCK_H2 + ty;


    int blockStartX = blockIdx.x*BLOCK_W - RADIUS;
    int blockStartY = blockIdx.y*BLOCK_H2 - RADIUS;

    const int blockSizeX = BLOCK_W + 2*RADIUS;
    const int blockSizeY = BLOCK_H2 + 2*RADIUS;

    //fill buffer
    for(int i = t; i < blockSizeX * blockSizeY; i += (BLOCK_W*BLOCK_H)){
        int x = i % blockSizeX;
        int y = i / blockSizeX;
        int gx = x + blockStartX;
        int gy = y + blockStartY;
        src.clampToEdge(gy,gx);
        buffer[y][x] = src(gy,gx);
    }

    __syncthreads();


    T *kernel = d_Kernel;

    for(int i = t; i < blockSizeX * BLOCK_H2; i += (BLOCK_W*BLOCK_H)){
        int x = i % blockSizeX;
        int y = i / blockSizeX;
        T sum = 0;
#pragma unroll
        for (int j=-RADIUS;j<=RADIUS;j++){
            int kernelIndex = j + RADIUS;
            sum += buffer[y + RADIUS + j][x] * kernel[kernelIndex];
        }
        buffer2[y][x] = sum;
    }

    __syncthreads();

    for(int i =0; i < Y_ELEMENTS; ++i){
        T sum = 0;
#pragma unroll
        for (int j=-RADIUS;j<=RADIUS;j++){
            int kernelIndex = j + RADIUS;
            sum += buffer2[ty][tx + RADIUS + j] * kernel[kernelIndex];
        }

        if(dst.inImage(yp,xp))
            dst(yp,xp) = sum;
        yp += BLOCK_H;
        ty += BLOCK_H;
    }
}

template<typename T, int RADIUS>
inline
void convolveOuterLinear(ImageView<T> src, ImageView<T> dst){
    int w = src.width;
    int h = src.height;

    const int BLOCK_W = 32;
    const int BLOCK_H = 16;
    const int Y_ELEMENTS = 2;
    dim3 blocks(
                Saiga::iDivUp(w, BLOCK_W ),
                Saiga::iDivUp(h, BLOCK_H * Y_ELEMENTS),
                1
                );

    //    dim3 blocks(Saiga::CUDA::getBlockCount(w, BLOCK_W), Saiga::CUDA::getBlockCount(h, BLOCK_H));
    dim3 threads(BLOCK_W, BLOCK_H);

    d_convolveOuterLinear<T,RADIUS,BLOCK_W,BLOCK_H,Y_ELEMENTS> <<<blocks, threads>>>(src,dst);
}


template<typename T, int RADIUS, unsigned int BLOCK_W, unsigned int BLOCK_H, unsigned int Y_ELEMENTS>
__global__ static
void d_convolveOuterHalo(ImageView<T> src, ImageView<T> dst)
{
    const unsigned int BLOCK_H2 = BLOCK_H * Y_ELEMENTS;
    const unsigned int WARPS_PER_BLOCK = BLOCK_W * BLOCK_H / 32; //16
    static_assert(WARPS_PER_BLOCK == 16, "warps per block wrong");



    //for radius = 4: elements = (32+8) * (16+8) = 960 = 3840
    __shared__ T buffer[BLOCK_H2 + 2*RADIUS][BLOCK_W + 2*RADIUS];
    //for radius = 4: elements = (32+8) * (16) = 640 = 2560
    __shared__ T buffer2[BLOCK_H2][BLOCK_W + 2*RADIUS];
    //total s mem per block = 6400
    //with 512 threads per block smem per sm: 25600 -> 100% occ


    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int t = tx + ty * BLOCK_W;
    const unsigned int warp_lane = t / 32;
    const unsigned int lane_id = t & 31;

    int xp = blockIdx.x*BLOCK_W + tx;
    int yp = blockIdx.y*BLOCK_H2 + ty;
    int x = xp;
    int y = yp;

    const unsigned int x_tile = blockIdx.x * BLOCK_W;
    //    const unsigned int y_tile = blockIdx.y * BLOCK_H2;

    int blockStartX = blockIdx.x*BLOCK_W - RADIUS;
    int blockStartY = blockIdx.y*BLOCK_H2 - RADIUS;

    const int blockSizeX = BLOCK_W + 2*RADIUS;
    //    const int blockSizeY = BLOCK_H2 + 2*RADIUS;

    //copy main data
    for(int i = 0; i < Y_ELEMENTS; ++i)
    {
        buffer[ty + i * BLOCK_H + RADIUS][tx + RADIUS]  = src.clampedRead(y + i * BLOCK_H,x);
    }

    //top and bottom halo
    if(warp_lane < 4)
    {
        const unsigned int num_warps = 4;
        for(int i = warp_lane; i < RADIUS; i+=num_warps)
        {
            buffer[i][lane_id + RADIUS]  =
                    src.clampedRead(blockStartY + i,x_tile + lane_id);

            buffer[BLOCK_H2 + RADIUS + i][lane_id + RADIUS]  =
                    src.clampedRead(blockStartY + BLOCK_H2 + RADIUS  + i,x_tile + lane_id);
        }
    }

    const unsigned int side_halo_rows_per_warp = 32 / RADIUS;

    int local_warp_id = lane_id / RADIUS;
    int local_lane_id = lane_id % RADIUS;

    //left halo
    if(warp_lane >= 4 && warp_lane < 10)
    {
        const unsigned int num_warps = 6;
        int wid = warp_lane - 4;
        int rows = BLOCK_H2 + 2 * RADIUS;

        for(int i = wid * side_halo_rows_per_warp + local_warp_id;i < rows; i += num_warps*side_halo_rows_per_warp)
        {
            if(local_warp_id < side_halo_rows_per_warp)
            {
                buffer[i][local_lane_id]  =
                        src.clampedRead(blockStartY + i,blockStartX + local_lane_id);
            }
        }
    }

    //right halo
    if(warp_lane >= 10 && warp_lane < 16)
    {
        const unsigned int num_warps = 6;
        int wid = warp_lane - 10;
        int rows = BLOCK_H2 + 2 * RADIUS;

        for(int i = wid * side_halo_rows_per_warp + local_warp_id;i < rows; i += num_warps*side_halo_rows_per_warp)
        {
            if(local_warp_id < side_halo_rows_per_warp)
            {
                buffer[i][local_lane_id + RADIUS + BLOCK_W]  =
                        src.clampedRead(blockStartY + i,blockStartX + local_lane_id + RADIUS + BLOCK_W);
            }
        }
    }

    __syncthreads();


    T *kernel = d_Kernel;

    for(int i = t; i < blockSizeX * BLOCK_H2; i += (BLOCK_W*BLOCK_H)){
        int x = i % blockSizeX;
        int y = i / blockSizeX;
        T sum = 0;
#pragma unroll
        for (int j=-RADIUS;j<=RADIUS;j++){
            int kernelIndex = j + RADIUS;
            sum += buffer[y + RADIUS + j][x] * kernel[kernelIndex];
        }
        buffer2[y][x] = sum;
    }

    __syncthreads();

    for(int i =0; i < Y_ELEMENTS; ++i){
        T sum = 0;
#pragma unroll
        for (int j=-RADIUS;j<=RADIUS;j++){
            int kernelIndex = j + RADIUS;
            sum += buffer2[ty][tx + RADIUS + j] * kernel[kernelIndex];
        }

        if(dst.inImage(yp,xp))
            dst(yp,xp) = sum;
        yp += BLOCK_H;
        ty += BLOCK_H;
    }
}

template<typename T, int RADIUS>
inline
void convolveOuterHalo(ImageView<T> src, ImageView<T> dst){
    int w = src.width;
    int h = src.height;

    const int BLOCK_W = 32;
    const int BLOCK_H = 16;
    const int Y_ELEMENTS = 2;
    dim3 blocks(
                Saiga::iDivUp(w, BLOCK_W ),
                Saiga::iDivUp(h, BLOCK_H * Y_ELEMENTS),
                1
                );

    //    dim3 blocks(Saiga::CUDA::getBlockCount(w, BLOCK_W), Saiga::CUDA::getBlockCount(h, BLOCK_H));
    dim3 threads(BLOCK_W, BLOCK_H);

    d_convolveOuterHalo<T,RADIUS,BLOCK_W,BLOCK_H,Y_ELEMENTS> <<<blocks, threads>>>(src,dst);
}


template<typename T, int RADIUS, unsigned int BLOCK_W, unsigned int BLOCK_H, unsigned int Y_ELEMENTS>
__global__ static
void d_convolveInner(ImageView<T> src, ImageView<T> dst)
{
    const unsigned int TILE_H = BLOCK_H;
    const unsigned int TILE_W = BLOCK_W;

    const unsigned int TILE_H2 = TILE_H * Y_ELEMENTS;
    const unsigned int tx = threadIdx.x;
    const unsigned int ty = threadIdx.y;
    //    int t = tx + ty * BLOCK_W;

    int x_tile = blockIdx.x * (TILE_W - 2 * RADIUS) - RADIUS;
    int y_tile = blockIdx.y * (TILE_H2 - 2 * RADIUS) - RADIUS;

    int x = x_tile + tx;
    int y = y_tile + ty;


    __shared__ T buffer[TILE_H2][TILE_W];
    __shared__ T buffer2[TILE_H2 - RADIUS * 2][TILE_W];



    //copy main data
    for(int i = 0; i < Y_ELEMENTS; ++i)
        buffer[ty + i * TILE_H][tx]  = src.clampedRead(y + i * TILE_H,x);



    __syncthreads();


    T *kernel = d_Kernel;

    //convolve along y axis
    //    if(ty > RADIUS && ty < TILE_H2 - RADIUS)
    //    {
    //        int oy = ty - RADIUS;

    for(int i = 0; i < Y_ELEMENTS; ++i)
    {
        //        int gx = x;
        //        int gy = y + i * TILE_H;
        int lx = tx;
        int ly = ty + i * TILE_H;

        if(ly < RADIUS || ly >= TILE_H2 - RADIUS)
            continue;

        T sum = 0;
#pragma unroll
        for (int j=-RADIUS;j<=RADIUS;j++)
        {
            int kernelIndex = j + RADIUS;
            sum += buffer[ly + j][lx] * kernel[kernelIndex];
        }
        buffer2[ly - RADIUS][lx] = sum;
    }



    __syncthreads();

    for(int i = 0; i < Y_ELEMENTS; ++i)
    {
        int gx = x;
        int gy = y + i * TILE_H;

        int lx = tx;
        int ly = ty + i * TILE_H;

        if(ly < RADIUS || ly >= TILE_H2 - RADIUS)
            continue;

        if(lx < RADIUS || lx >= TILE_W - RADIUS)
            continue;

        T sum = 0;
#pragma unroll
        for (int j=-RADIUS;j<=RADIUS;j++)
        {
            int kernelIndex = j + RADIUS;
            sum += buffer2[ly - RADIUS][lx + j] * kernel[kernelIndex];
        }

        //        if(dst.inImage(gx,gy))
        //            dst(g,yp) = sum;
        dst.clampedWrite(gy,gx,sum);
    }



#if 0

    for(int i =0; i < Y_ELEMENTS; ++i){
        T sum = 0;
#pragma unroll
        for (int j=-RADIUS;j<=RADIUS;j++){
            int kernelIndex = j + RADIUS;
            sum += buffer2[ty][tx + RADIUS + j] * kernel[kernelIndex];
        }

        if(dst.inImage(xp,yp))
            dst(xp,yp) = sum;
        yp += BLOCK_H;
        ty += BLOCK_H;
    }
#endif
}

template<typename T, int RADIUS, bool LOW_OCC = false>
inline
void convolveInner(ImageView<T> src, ImageView<T> dst){
    int w = src.width;
    int h = src.height;


    const int BLOCK_W = LOW_OCC ? 64 : 32;
    const int BLOCK_H = LOW_OCC ? 8 : 16;
    const int Y_ELEMENTS = LOW_OCC ? 4 : 2;
    dim3 blocks(
                Saiga::iDivUp(w, BLOCK_W - 2 * RADIUS),
                Saiga::iDivUp(h, BLOCK_H * Y_ELEMENTS - 2 * RADIUS),
                1
                );

    //    dim3 blocks(Saiga::CUDA::getBlockCount(w, BLOCK_W), Saiga::CUDA::getBlockCount(h, BLOCK_H));
    dim3 threads(BLOCK_W, BLOCK_H);

    d_convolveInner<T,RADIUS,BLOCK_W,BLOCK_H,Y_ELEMENTS> <<<blocks, threads>>>(src,dst);
}



template<typename T, int RADIUS, unsigned int BLOCK_W, unsigned int BLOCK_H, unsigned int Y_ELEMENTS>
__global__ static
void d_convolveInnerShuffle(ImageView<T> src, ImageView<T> dst)
{
    const unsigned int TILE_H = BLOCK_H;
    const unsigned int TILE_W = BLOCK_W;

    const unsigned int TILE_H2 = TILE_H * Y_ELEMENTS;
    const unsigned int tx = threadIdx.x;
    const unsigned int ty = threadIdx.y;
    //    int t = tx + ty * BLOCK_W;

    unsigned int lane_id = threadIdx.x % 32;

    int x_tile = blockIdx.x * (TILE_W - 2 * RADIUS) - RADIUS;
    int y_tile = blockIdx.y * (TILE_H2 - 2 * RADIUS) - RADIUS;

    int x = x_tile + tx;
    int y = y_tile + ty;




    //    __shared__ T buffer[TILE_H2][TILE_W];
    //    __shared__ T buffer2[TILE_H2][TILE_W - RADIUS * 2 + 1];
    __shared__ T buffer2[TILE_H2][TILE_W - RADIUS * 2];
    //    __shared__ T buffer2[TILE_W - RADIUS * 2][TILE_H2];


    T localElements[Y_ELEMENTS];
    for(int i = 0; i < Y_ELEMENTS; ++i)
    {
        localElements[i] = src.clampedRead(y + i * TILE_H,x);
    }

    //conv row

    T *kernel = d_Kernel;


    for(int i = 0; i < Y_ELEMENTS; ++i)
    {
        int lx = tx;
        int ly = ty + i * TILE_H;
        T sum = 0;
#pragma unroll
        for (int j=-RADIUS;j<=RADIUS;j++)
        {
            int kernelIndex = j + RADIUS;
            auto value =  shfl(localElements[i],lane_id + j);

            sum += value * kernel[kernelIndex];
        }

        if(lx < RADIUS || lx >= TILE_W - RADIUS)
            continue;

        buffer2[ly][lx- RADIUS] = sum;
        //        buffer2[lx- RADIUS][ly] = sum;
    }



    __syncthreads();

    //conv col

    for(int i = 0; i < Y_ELEMENTS; ++i)
    {
        int gx = x;
        int gy = y + i * TILE_H;

        int lx = tx;
        int ly = ty + i * TILE_H;

        if(ly < RADIUS || ly >= TILE_H2 - RADIUS)
            continue;

        if(lx < RADIUS || lx >= TILE_W - RADIUS)
            continue;

        T sum = 0;
#if 1
#pragma unroll
        for (int j=-RADIUS;j<=RADIUS;j++)
        {
            int kernelIndex = j + RADIUS;
            auto value = buffer2[ly + j][lx - RADIUS];
            //            auto value = buffer2[lx - RADIUS][ly + j];
            sum +=  value * kernel[kernelIndex];
        }
#endif
        dst.clampedWrite(gy,gx,sum);
    }


}



template<typename T, int RADIUS, unsigned int BLOCK_W, unsigned int BLOCK_H, unsigned int X_ELEMENTS, unsigned int Y_ELEMENTS>
__global__ static
void d_convolveInnerShuffle2(ImageView<T> src, ImageView<T> dst)
{
    const unsigned int TILE_W = BLOCK_W;
    const unsigned int TILE_H = BLOCK_H;

    const unsigned int TILE_W2 = TILE_W * X_ELEMENTS;
    const unsigned int TILE_H2 = TILE_H * Y_ELEMENTS;
    const unsigned int tx = threadIdx.x;
    const unsigned int ty = threadIdx.y;
    //    int t = tx + ty * BLOCK_W;

    using VectorType = int2;

    unsigned int lane_id = threadIdx.x % 32;

    //start position of tile
    int x_tile = blockIdx.x * (TILE_W2 - 2 * RADIUS) - RADIUS;
    int y_tile = blockIdx.y * (TILE_H2 - 2 * RADIUS) - RADIUS;

    //global position of thread
    int x = x_tile + tx * X_ELEMENTS;
    int y = y_tile + ty;


    T *kernel = d_Kernel;


    __shared__ VectorType buffer2[TILE_H2][TILE_W - RADIUS / X_ELEMENTS * 2];


    VectorType localElements[Y_ELEMENTS][6];

//#pragma unroll(1)
    for(int i = 0; i < Y_ELEMENTS; ++i)
    {
        int rowId = y + i * TILE_H;
        rowId = min(rowId,src.height-1);
        rowId = max(0,rowId);

        int colId = max(0,x);
        colId = min(colId,src.width - 2);



        T* row = src.rowPtr(rowId);
        T* elem = row + colId;

        VectorType& myValue = localElements[i][RADIUS / X_ELEMENTS];


        myValue = reinterpret_cast<VectorType*>(elem)[0];


        //shuffle left
        for(int j = -2; j <= -1 ; ++j)
        {
            localElements[i][j + 2] =  shfl(myValue,lane_id + j);
        }

        //shuffle right
        for(int j = 1; j <= 2 ; ++j)
        {
            localElements[i][j + 2] =  shfl(myValue,lane_id + j);
        }


        T* localElementsT = reinterpret_cast<T*>(localElements[i]);
        int offsetA = RADIUS;
        int offsetB = RADIUS + 1;


        T sum[2];
        for(int j = 0; j < 2; ++j)
        {
            sum[j] = 0;
        }

        for (int j=-RADIUS;j<=RADIUS;j++)
        {
            int kernelIndex = j + RADIUS;
            T kernelValue = kernel[kernelIndex];
//            kernelValue = 1.0f;

            T valueA =  localElementsT[offsetA + j];
            T valueB =  localElementsT[offsetB + j];

            sum[0] += valueA * kernelValue;
            sum[1] += valueB * kernelValue;
//            sum[0] += 1;
//            sum[1] += 1;
        }

//        myValue = reinterpret_cast<VectorType*>(sum)[0];

//                if(x < 5 && rowId == 44)
//                {
//                    printf("%d %d %d %d %f %f\n",x,y,rowId,colId, sum[0], sum[1]);
//                }

        int lx = tx;
        int ly = ty + i * TILE_H;
        if(lx < RADIUS / 2 || lx >= TILE_W - RADIUS / 2)
            continue;

        buffer2[ly][lx - RADIUS / 2] = reinterpret_cast<VectorType*>(sum)[0];
    }


    __syncthreads();





    for(int i = 0; i < Y_ELEMENTS; ++i)
    {
        int rowId = y + i * TILE_H;
        rowId = min(rowId,src.height-1);
        rowId = max(0,rowId);

        int colId = max(0,x);
        colId = min(colId,src.width - 2);

        int lx = tx;
        int ly = ty + i * TILE_H;

        if(lx < RADIUS / 2 || lx >= TILE_W - RADIUS / 2)
            continue;
        if(ly < RADIUS || ly >= TILE_H2 - RADIUS)
            continue;

        T* row = dst.rowPtr(rowId);
        T* elem = row + colId;




        T sum[2];
        for(int j = 0; j < 2; ++j)
        {
            sum[j] = 0;
        }

        for (int j=-RADIUS;j<=RADIUS;j++)
//        for (int j=0;j<=0;j++)
        {
            int kernelIndex = j + RADIUS;
            T kernelValue = kernel[kernelIndex];


            VectorType valueV =  buffer2[ly][lx - RADIUS / X_ELEMENTS];

            sum[0] += reinterpret_cast<T*>(&valueV)[0] * kernelValue;
            sum[1] += reinterpret_cast<T*>(&valueV)[1] * kernelValue;
        }

//        VectorType& myValue = localElements[i][RADIUS / X_ELEMENTS];
//        reinterpret_cast<VectorType*>(elem)[0] = reinterpret_cast<VectorType*>(&myValue)[0];
        reinterpret_cast<VectorType*>(elem)[0] = reinterpret_cast<VectorType*>(sum)[0];
    }

#if 0
    //conv row

    T *kernel = d_Kernel;


    for(int i = 0; i < Y_ELEMENTS; ++i)
    {
        int lx = tx;
        int ly = ty + i * TILE_H;
        T sum = 0;
#pragma unroll
        for (int j=-RADIUS;j<=RADIUS;j++)
        {
            int kernelIndex = j + RADIUS;
            auto value =  shfl(localElements[i],lane_id + j);

            sum += value * kernel[kernelIndex];
        }

        if(lx < RADIUS || lx >= TILE_W - RADIUS)
            continue;

        buffer2[ly][lx- RADIUS] = sum;
        //        buffer2[lx- RADIUS][ly] = sum;
    }



    __syncthreads();

    //conv col

    for(int i = 0; i < Y_ELEMENTS; ++i)
    {
        int gx = x;
        int gy = y + i * TILE_H;

        int lx = tx;
        int ly = ty + i * TILE_H;

        if(ly < RADIUS || ly >= TILE_H2 - RADIUS)
            continue;

        if(lx < RADIUS || lx >= TILE_W - RADIUS)
            continue;

        T sum = 0;
#if 1
#pragma unroll
        for (int j=-RADIUS;j<=RADIUS;j++)
        {
            int kernelIndex = j + RADIUS;
            auto value = buffer2[ly + j][lx - RADIUS];
            //            auto value = buffer2[lx - RADIUS][ly + j];
            sum +=  value * kernel[kernelIndex];
        }
#endif
        //        dst.clampedWrite(gy,gx,sum);
    }
#endif

}

template<typename T, int RADIUS, bool LOW_OCC = false>
inline
void convolveInnerShuffle(ImageView<T> src, ImageView<T> dst){
    int w = src.width;
    int h = src.height;


    const int BLOCK_W = 32;
    const int BLOCK_H = 16;

    const int X_ELEMENTS = 2;
    const int Y_ELEMENTS = 4;
    dim3 blocks(
                Saiga::iDivUp(w, BLOCK_W * X_ELEMENTS - 2 * RADIUS),
                Saiga::iDivUp(h, BLOCK_H * Y_ELEMENTS - 2 * RADIUS),
                1
                );

//    blocks.y = 4;

    //    dim3 blocks(Saiga::CUDA::getBlockCount(w, BLOCK_W), Saiga::CUDA::getBlockCount(h, BLOCK_H));
    dim3 threads(BLOCK_W, BLOCK_H);


      hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(d_convolveInnerShuffle2<T),RADIUS,BLOCK_W,BLOCK_H,X_ELEMENTS,Y_ELEMENTS>,hipSharedMemBankSizeEightByte);
//    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(d_convolveInnerShuffle2<T),RADIUS,BLOCK_W,BLOCK_H,X_ELEMENTS,Y_ELEMENTS>,hipSharedMemBankSizeFourByte);

    d_convolveInnerShuffle2<T,RADIUS,BLOCK_W,BLOCK_H,X_ELEMENTS,Y_ELEMENTS> <<<blocks, threads>>>(src,dst);
}

void convolveSinglePassSeparateOuterLinear(ImageView<float> src, ImageView<float> dst, Saiga::array_view<float> kernel, int radius){
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel), kernel.data(), kernel.size()*sizeof(float),0,hipMemcpyDeviceToDevice));
    switch (radius){
    case 1: CUDA::convolveOuterLinear<float,1>(src,dst); break;
    case 2: CUDA::convolveOuterLinear<float,2>(src,dst); break;
    case 3: CUDA::convolveOuterLinear<float,3>(src,dst); break;
    case 4: CUDA::convolveOuterLinear<float,4>(src,dst); break;
    case 5: CUDA::convolveOuterLinear<float,5>(src,dst); break;
    case 6: CUDA::convolveOuterLinear<float,6>(src,dst); break;
    case 7: CUDA::convolveOuterLinear<float,7>(src,dst); break;
    case 8: CUDA::convolveOuterLinear<float,8>(src,dst); break;
    case 9: CUDA::convolveOuterLinear<float,9>(src,dst); break;
    }
}

void convolveSinglePassSeparateOuterHalo(ImageView<float> src, ImageView<float> dst, Saiga::array_view<float> kernel, int radius){
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel), kernel.data(), kernel.size()*sizeof(float),0,hipMemcpyDeviceToDevice));
    switch (radius){
    case 1: CUDA::convolveOuterHalo<float,1>(src,dst); break;
    case 2: CUDA::convolveOuterHalo<float,2>(src,dst); break;
    case 3: CUDA::convolveOuterHalo<float,3>(src,dst); break;
    case 4: CUDA::convolveOuterHalo<float,4>(src,dst); break;
    case 5: CUDA::convolveOuterHalo<float,5>(src,dst); break;
    case 6: CUDA::convolveOuterHalo<float,6>(src,dst); break;
    case 7: CUDA::convolveOuterHalo<float,7>(src,dst); break;
    case 8: CUDA::convolveOuterHalo<float,8>(src,dst); break;
    }
}

void convolveSinglePassSeparateInner(ImageView<float> src, ImageView<float> dst, Saiga::array_view<float> kernel, int radius){
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel), kernel.data(), kernel.size()*sizeof(float),0,hipMemcpyDeviceToDevice));
    switch (radius){
    case 1: CUDA::convolveInner<float,1>(src,dst); break;
    case 2: CUDA::convolveInner<float,2>(src,dst); break;
    case 3: CUDA::convolveInner<float,3>(src,dst); break;
    case 4: CUDA::convolveInner<float,4>(src,dst); break;
    case 5: CUDA::convolveInner<float,5>(src,dst); break;
    case 6: CUDA::convolveInner<float,6>(src,dst); break;
    case 7: CUDA::convolveInner<float,7>(src,dst); break;
    case 8: CUDA::convolveInner<float,8>(src,dst); break;
    }
}


void convolveSinglePassSeparateInner75(ImageView<float> src, ImageView<float> dst, Saiga::array_view<float> kernel, int radius){
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel), kernel.data(), kernel.size()*sizeof(float),0,hipMemcpyDeviceToDevice));
    switch (radius){
    case 1: CUDA::convolveInner<float,1,true>(src,dst); break;
    case 2: CUDA::convolveInner<float,2,true>(src,dst); break;
    case 3: CUDA::convolveInner<float,3,true>(src,dst); break;
    case 4: CUDA::convolveInner<float,4,true>(src,dst); break;
    case 5: CUDA::convolveInner<float,5,true>(src,dst); break;
    case 6: CUDA::convolveInner<float,6,true>(src,dst); break;
    case 7: CUDA::convolveInner<float,7,true>(src,dst); break;
    case 8: CUDA::convolveInner<float,8,true>(src,dst); break;
    }
}


void convolveSinglePassSeparateInnerShuffle(ImageView<float> src, ImageView<float> dst, Saiga::array_view<float> kernel, int radius){
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_Kernel), kernel.data(), kernel.size()*sizeof(float),0,hipMemcpyDeviceToDevice));
    switch (radius){
    //    case 1: CUDA::convolveInnerShuffle<float,1,true>(src,dst); break;
    //    case 2: CUDA::convolveInnerShuffle<float,2,true>(src,dst); break;
    //    case 3: CUDA::convolveInnerShuffle<float,3,true>(src,dst); break;
    //    case 4: CUDA::convolveInnerShuffle<float,4,true>(src,dst); break;
    //    case 5: CUDA::convolveInnerShuffle<float,5,true>(src,dst); break;
    //    case 6: CUDA::convolveInnerShuffle<float,6,true>(src,dst); break;
    //    case 7: CUDA::convolveInnerShuffle<float,7,true>(src,dst); break;
    //    case 8: CUDA::convolveInnerShuffle<float,8,true>(src,dst); break;
    case 4: CUDA::convolveInnerShuffle<float,4,true>(src,dst); break;
    }
}

}
}
