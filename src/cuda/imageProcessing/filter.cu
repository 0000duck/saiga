#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2017 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "saiga/cuda/imageProcessing/filter.h"
#include "saiga/cuda/imageProcessing/convolution.h"

namespace Saiga {
namespace CUDA {

thrust::device_vector<float>  createGaussianBlurKernel(int radius, float sigma){
    SAIGA_ASSERT(radius <= MAX_RADIUS && radius > 0);
    const int ELEMENTS = radius * 2 + 1;
    thrust::host_vector<float> kernel(ELEMENTS);
    float kernelSum = 0.0f;
    float ivar2 = 1.0f/(2.0f*sigma*sigma);
    for (int j=-radius;j<=radius;j++) {
        kernel[j+radius] = (float)expf(-(double)j*j*ivar2);
        kernelSum += kernel[j+radius];
    }
    for (int j=-radius;j<=radius;j++)
        kernel[j+radius] /= kernelSum;
    return thrust::device_vector<float>(kernel);
}


void applyFilterSeparate(ImageView<float> src, ImageView<float> dst, ImageView<float> tmp, array_view<float> kernelRow, array_view<float> kernelCol){
    convolveRow(src,tmp,kernelRow,kernelRow.size() / 2);
    convolveCol(tmp,dst,kernelCol,kernelCol.size() / 2);
}

void applyFilterSeparateSinglePass(ImageView<float> src, ImageView<float> dst, array_view<float> kernel){
    convolveSinglePassSeparate(src,dst,kernel,kernel.size()/2);
}


}
}


