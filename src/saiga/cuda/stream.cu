/**
 * Copyright (c) 2017 Darius Rückert
 * Licensed under the MIT License.
 * See LICENSE file for more information.
 */

#include "stream.h"


namespace Saiga
{
namespace CUDA
{
CudaStream::CudaStream() { hipStreamCreate(&stream); }

CudaStream::~CudaStream() { hipStreamDestroy(stream); }

void CudaStream::waitForEvent(hipEvent_t event) { hipStreamWaitEvent(stream, event, 0); }

hipStream_t CudaStream::legacyStream() { return hipStreamLegacy; }

hipStream_t CudaStream::perThreadStream() { return hipStreamPerThread; }

Saiga::CUDA::CudaStream::operator hipStream_t() const { return stream; }
}  // namespace CUDA
}  // namespace Saiga
